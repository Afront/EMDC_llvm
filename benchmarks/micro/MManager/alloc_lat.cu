#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

int main(int argc, char* argv[]){
	hipFreeArray(0);
	
	float* arr;
	unsigned long long int size=4096; // 0
	hipError_t err;
	unsigned long long int limit = 34359738368;

	unsigned int arg = atoi(argv[1]);

	size <<= arg;

	printf("size: %llu\n",size);

	err = hipMalloc((void**)&arr, size);
	if(err != hipSuccess){
		printf("err: %s\n",hipGetErrorString(err));
	}


	return 0;
}
