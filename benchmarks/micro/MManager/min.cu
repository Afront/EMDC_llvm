#include <hip/hip_runtime.h>
#include <iostream>
#include <nvml.h>
#include <stdio.h>

#include <unistd.h>

using STACK_SIZE_T = size_t;
using DATA_TYPE = float;

#define ALLOC_SIZE 1024
#define MB 1048576
#define KB 1024

__global__ void foo_kernel(){
	printf("Hello\n");	
}

int main(){
	hipFreeArray(0);
	STACK_SIZE_T stack_size, fifo_size, heap_size, sync_depth, pend_cnt;	

#ifdef NVML
	nvmlDevice_t dev_id_nvml;
	nvmlReturn_t ret_nvml;
	nvmlInit();

	ret_nvml = nvmlDeviceGetHandleByIndex(0,&dev_id_nvml);
	nvmlMemory_t mem_info;
#endif

	DATA_TYPE *arr;
	hipMalloc((void**)&arr, ALLOC_SIZE);
	hipFree(arr);

	//system("nvidia-smi");
	hipDeviceSetLimit(hipLimitStackSize, 16);
	hipDeviceGetLimit(&stack_size , hipLimitStackSize);
	std::cout<<"Limit Stack to size "<<stack_size<<"\n";
#ifdef NVML
	ret_nvml = nvmlDeviceGetMemoryInfo(dev_id_nvml, &mem_info);
	std::cout<<"GMem used: "<<mem_info.used/KB<<"(KB)\n";
	//system("nvidia-smi");
#endif

	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth,1);
	hipDeviceGetLimit(&sync_depth, cudaLimitDevRuntimeSyncDepth);
	std::cout<<"Limit sycn depth to "<<sync_depth<<"\n";

	hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 4096);
	hipDeviceGetLimit(&pend_cnt, cudaLimitDevRuntimePendingLaunchCount);
	std::cout<<"Limit pending count to "<<pend_cnt<<"\n";

	hipDeviceSetLimit(hipLimitPrintfFifoSize, 32);
	hipDeviceGetLimit(&fifo_size, hipLimitPrintfFifoSize);
	std::cout<<"Limit printf FIFO to size "<<(float)fifo_size/KB<<"\n";
#ifdef NVML	
	ret_nvml = nvmlDeviceGetMemoryInfo(dev_id_nvml, &mem_info);
	std::cout<<"GMem used: "<<mem_info.used/KB<<"(KB)\n";
#endif

	hipDeviceSetLimit(hipLimitMallocHeapSize, 1024);
	hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
	std::cout<<"Limit malloc heap to size "<<heap_size/KB<<"\n";
#ifdef NVML
	ret_nvml = nvmlDeviceGetMemoryInfo(dev_id_nvml, &mem_info);
	std::cout<<"GMem used: "<<(float)mem_info.used/KB<<"(KB)\n";
#endif
	foo_kernel<<<1,1>>>();
	hipDeviceSynchronize();
#ifdef NVML
	nvmlShutdown();
#endif

	system("nvidia-smi");

//	sleep(10);
	return 0;	
}
