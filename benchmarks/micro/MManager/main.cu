#include <hip/hip_runtime.h>
#include <iostream>

using STACK_SIZE_T = size_t;
using DATA_TYPE = float;

#define ALLOC_SIZE 1024

int main(){
	hipFreeArray(0);
	
	STACK_SIZE_T stack_size;
	hipDeviceGetLimit(&stack_size, hipLimitStackSize);
	std::cout<<"Stack size limit : "<<stack_size<<"\n";

	DATA_TYPE *arr;
	hipMalloc((void**)&arr, ALLOC_SIZE);
	std::cout<<"After cudaMalloc(default stack size)\n";
	system("nvidia-smi");
	hipFree(arr);

	std::cout<<"After free...\n";
	system("nvidia-smi");
	hipDeviceSetLimit(hipLimitStackSize, stack_size/2);
	hipDeviceGetLimit(&stack_size, hipLimitStackSize);
	std::cout<<"Stack size limit : "<<stack_size<<"\n";

	hipMalloc((void**)&arr, ALLOC_SIZE);
	std::cout<<"After cudaMalloc(stack size/=2)\n";
	system("nvidia-smi");
	hipFree(arr);


	return 0;	
}
